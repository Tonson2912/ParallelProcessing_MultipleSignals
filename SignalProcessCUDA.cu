#include "hip/hip_runtime.h"
#include"func_header.h"
#include"SignalProcessCUDA.cuh"


class SignalProcessCUDA:public signal_process_model<float>
{
public:
	SignalProcessCUDA(unsigned int height,unsigned int width, unsigned int num,unsigned int patch_num=1,unsigned int stream_num=1) :_height(height),_width(width),_num(num),_patch_num(patch_num),_stream_num(stream_num) {
		_length = _height * _width * _num;
		_patch_size = iDivide(_height * _width, _patch_num);
	}
	void set_frequency(int low_freq, int high_freq, FilterMode fliter_mode)
	{
		_low_freq = low_freq;
		_high_freq = high_freq;
		_fliter_mode = fliter_mode;
	}
	void compute_gpu(float* datalines_src,float* signal_real_dst,float* signal_abs_dst, bool is_need_trans)
	{
		float time1 = 0;
		cout << "Start timing the gpu run time--compute_gpu" << endl;
		hipEvent_t start, end;
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);
		compute(datalines_src, signal_real_dst, signal_abs_dst, is_need_trans);
		hipEventRecord(end, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(end);
		hipEventElapsedTime(&time1, start, end);
		hipEventDestroy(start);
		hipEventDestroy(end);
		cout << "End of the timing" << endl;
		cout << "GPU: " << time1 << "ms" << endl << endl << endl;
	}
	void reset()
	{
		return;
	}
	void del_object() {
		delete this;
	}
private:
	~SignalProcessCUDA()
	{
		cuda_free();
	}
	inline int iDivide(int a, int b) {
		return a % b != 0 ? a / b + 1 : a / b;
	}
	void cuda_init(int patch_size, bool is_malloc_abs_dst, bool is_malloc_real_dst);
	void compute(float* datalines_src, float* signal_dst, float* envelope_dst, bool is_need_trans);
	void cuda_free();
	FilterMode _fliter_mode;
	unsigned int _length;
	unsigned int _height;
	unsigned int _width;
	unsigned int _num;
	int _low_freq;
	int _high_freq;
	//gpu�豸�ڴ�
	float* _d_dataline = nullptr;
	float* _d_signal_real = nullptr;
	float* _d_signal_abs = nullptr;
	hipfftComplex* _d_dataline_comp = nullptr;
	hipfftComplex* _d_dft_result_comp = nullptr;
	hipfftComplex* _d_idft_result_comp = nullptr;
	//�������������
	unsigned int _patch_num;//��������
	unsigned int _patch_size;//ÿһ���Ĵ�С��ģ��_patch_size���źţ�ÿ���ź�_num���㣩
	unsigned int _stream_num;
};

void SignalProcessCUDA::cuda_init(int patch_size,bool is_malloc_abs_dst,bool is_malloc_real_dst)
{
	int patch_length = patch_size * _num;
	//�����豸�ڴ�
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dataline, sizeof(float) * patch_length));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dataline_comp, sizeof(hipfftComplex) * patch_length));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dft_result_comp, sizeof(hipfftComplex) * patch_length));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_idft_result_comp, sizeof(hipfftComplex) * patch_length));
	if (is_malloc_abs_dst)RUNTIME_CUDA_ERROR(hipMalloc(&_d_signal_abs, sizeof(float) * patch_length));
	if (is_malloc_real_dst)RUNTIME_CUDA_ERROR(hipMalloc(&_d_signal_real, sizeof(float) * patch_length));
}

void SignalProcessCUDA::compute(float* datalines_src, float* signal_real_dst, float* signal_abs_dst, bool is_need_trans)
{
	auto streams = new hipStream_t[_stream_num];
	auto fftPlanFwd = new hipfftHandle[_stream_num];
	//����������
	for (int k = 0; k < _stream_num; ++k)
	{
		RUNTIME_CUDA_ERROR(hipStreamCreate(streams + k));
	}
	RUNTIME_CUDA_ERROR(hipGetLastError());
	cuda_init(_patch_size, signal_abs_dst != nullptr, signal_real_dst != nullptr);
	for (int i = 0; i < _patch_num; ++i)
	{
		//��ǰ��������Ĵ�С�ͳ��ȡ�
		unsigned int patch_size = min(_height * _width - i * _patch_size, _patch_size);
		unsigned int patch_length = patch_size * _num;
		//ԭʼ�����ڵ�ǰ����������Ծ��ָ���ַ��С�ͳ��ȡ�
		unsigned int jump_patch_size = i * _patch_size;
		unsigned int jump_patch_length = i * _patch_size * _num;

		//�����豸�ڴ�
		//if (patch_size != _patch_size)cuda_init(patch_size, signal_abs_dst != nullptr, signal_real_dst != nullptr);

		//���������
		unsigned int chunk_size = iDivide(patch_size, _stream_num);
		auto streams_size = new unsigned int[_stream_num];
		for (int stream = 0; stream < _stream_num; ++stream)
		{
			//��ǰ��������Ĵ�С�ͳ��ȡ�
			streams_size[stream] = min(patch_size - stream * chunk_size, chunk_size);
		}
		for (int stream = 0; stream < _stream_num; ++stream)
		{
			//��ǰ��������Ĵ�С�ͳ��ȡ�
			unsigned int stream_size = streams_size[stream];
			unsigned int stream_length = stream_size * _num;
			//ԭʼ�����ڵ�ǰ����������Ծ��ָ���ַ��С�ͳ��ȡ�
			unsigned int jump_stream_size = stream * chunk_size;
			unsigned int jump_stream_length = stream * chunk_size * _num;
			//FFT
			int fft_patch_num = stream_size;
			int n[1] = { _num };
			int inembed[2] = { _num,fft_patch_num };
			int onembed[2] = { _num,fft_patch_num };
			CUFFT_CUDA_ERROR(hipfftPlanMany(fftPlanFwd + stream, 1, n, inembed, 1, _num, onembed, 1, _num, HIPFFT_C2C, fft_patch_num));
			CUFFT_CUDA_ERROR(hipfftSetStream(fftPlanFwd[stream], streams[stream]));


			//�������ڴ濽�����豸�ڴ�
			if (_patch_num == 1)
			{
				if (_stream_num == 1)//ȫ������
				{
					RUNTIME_CUDA_ERROR(hipMemcpyAsync(_d_dataline + jump_stream_length, datalines_src + jump_stream_length, sizeof(float) * stream_length, hipMemcpyHostToDevice, streams[stream]));
				}
				else
				{
					if (is_need_trans)
					{
						for (int j = 0; j < _num; ++j)//һ֡һ֡���ƣ���Ϊ�����Ǵ�ͼ��ά�Ƚ��зָ�
						{
							RUNTIME_CUDA_ERROR(hipMemcpyAsync(_d_dataline + jump_stream_length + j * stream_size, datalines_src + jump_stream_size + j * _height * _width, sizeof(float) * stream_size, hipMemcpyHostToDevice, streams[stream]));
						}
					}
					else
					{
						RUNTIME_CUDA_ERROR(hipMemcpyAsync(_d_dataline + jump_stream_length, datalines_src + jump_stream_length, sizeof(float) * stream_length, hipMemcpyHostToDevice, streams[stream]));
					}
				}
			}
			else
			{
				if (is_need_trans)
				{
					for (int j = 0; j < _num; ++j)//һ֡һ֡���ƣ���Ϊ�����Ǵ�ͼ��ά�Ƚ��зָ�
					{
						//RUNTIME_CUDA_ERROR(hipMemcpy(_d_dataline + j * patch_size, datalines_src + jump_patch_size + j * _height * _width, sizeof(float) * patch_size, hipMemcpyHostToDevice));
						RUNTIME_CUDA_ERROR(hipMemcpyAsync(_d_dataline + jump_stream_length + j * stream_size, datalines_src + jump_patch_size + jump_stream_size + j * _height * _width, sizeof(float) * stream_size, hipMemcpyHostToDevice, streams[stream]));
					}
				}
				else
				{
					//RUNTIME_CUDA_ERROR(hipMemcpy(_d_dataline, datalines_src + jump_patch_length, sizeof(float) * patch_length, hipMemcpyHostToDevice));
					RUNTIME_CUDA_ERROR(hipMemcpyAsync(_d_dataline + jump_stream_length, datalines_src + jump_patch_length + jump_stream_length, sizeof(float) * stream_length, hipMemcpyHostToDevice, streams[stream]));
				}
			}
		}
		for(int stream = 0; stream < _stream_num; stream++)
		{
			//��ǰ��������Ĵ�С�ͳ��ȡ�
			unsigned int stream_size = streams_size[stream];
			unsigned int stream_length = stream_size * _num;
			//ԭʼ�����ڵ�ǰ����������Ծ��ָ���ַ��С�ͳ��ȡ�
			unsigned int jump_stream_length = stream * chunk_size * _num;
			//ת��ά�Ȳ��ҽ���R2Z
			dim3 block(1, 512);
			dim3 grid(stream_size, iDivide(_num, block.y));
			if (is_need_trans)//44ms
			{
				//ת��ά�ȣ�����ά��Ϊ��һά��
				trans_dim_R2C_kernel << <grid, block, 0, streams[stream] >> > (_d_dataline + jump_stream_length, _d_dataline_comp + jump_stream_length, _num);
			}
			else
			{
				cufft_R2C_kernel << <iDivide(stream_length, 256), 256,0, streams[stream] >> > (_d_dataline + jump_stream_length, _d_dataline_comp + jump_stream_length, stream_length);
			}
			RUNTIME_CUDA_ERROR(hipGetLastError());

			//����Ҷ�仯Z2Z
			CUFFT_CUDA_ERROR(hipfftExecC2C(fftPlanFwd[stream], _d_dataline_comp + jump_stream_length, _d_dft_result_comp + jump_stream_length, HIPFFT_FORWARD));//14ms

			//��ͨ��Χ�ڻ��ͨ��Χ��
			if (_fliter_mode == FilterMode::NON_BANDPASS)
			{
				low_high_frequency_value2zero_kernel << <grid, block, 0, streams[stream] >> > (_d_dft_result_comp + jump_stream_length, _low_freq, _high_freq, _num);
			}
			else
			{
				none_low_high_frequency_value2zero_kernel << <grid, block,0, streams[stream] >> > (_d_dft_result_comp + jump_stream_length, _low_freq, _high_freq, _num);
			}
			RUNTIME_CUDA_ERROR(hipGetLastError());


			//�渵��Ҷ�仯Z2Z
			CUFFT_CUDA_ERROR(hipfftExecC2C(fftPlanFwd[stream], _d_dft_result_comp + jump_stream_length, _d_idft_result_comp + jump_stream_length, HIPFFT_BACKWARD));//14ms



			//ifft��ɢ����Ҷ�仯֮����Ҫ����N
			cufft_divide_N_kernel << <iDivide(stream_length, 256), 256,0, streams[stream] >> > (_d_idft_result_comp + jump_stream_length, _num, stream_length);//16

			//ȡʵ��(�����ź�)
			if (signal_real_dst != nullptr)
			{
				cufft_comp_real_kernel << <iDivide(stream_length, 256), 256, 0, streams[stream] >> > (_d_idft_result_comp + jump_stream_length, _d_signal_real + jump_stream_length, stream_length);//11
				RUNTIME_CUDA_ERROR(hipGetLastError());
			}
			//ȡ����ֵ(����)
			if (signal_abs_dst != nullptr)
			{
				cufft_comp_asb_kernel << <iDivide(stream_length, 256), 256, 0, streams[stream] >> > (_d_idft_result_comp + jump_stream_length, _d_signal_abs + jump_stream_length, stream_length);//65
				RUNTIME_CUDA_ERROR(hipGetLastError());
			}
		}
		for (int stream = 0; stream < _stream_num; stream++)
		{
			//��ǰ��������Ĵ�С�ͳ��ȡ�
			unsigned int stream_length = streams_size[stream] * _num;
			//ԭʼ�����ڵ�ǰ����������Ծ��ָ���ַ��С�ͳ��ȡ�
			unsigned int jump_stream_length = stream * chunk_size * _num;
			//ȡʵ��(�����ź�)
			if (signal_real_dst != nullptr)
			{
				//���豸�ڴ濽���������ڴ�
				RUNTIME_CUDA_ERROR(hipMemcpyAsync(signal_real_dst + jump_patch_length + jump_stream_length, _d_signal_real + jump_stream_length, sizeof(float)* stream_length, hipMemcpyDeviceToHost, streams[stream]));
			}
			//ȡ����ֵ(����)
			if (signal_abs_dst != nullptr)
			{
				//���豸�ڴ濽���������ڴ�
				RUNTIME_CUDA_ERROR(hipMemcpyAsync(signal_abs_dst + jump_patch_length + jump_stream_length, _d_signal_abs + jump_stream_length, sizeof(float)* stream_length, hipMemcpyDeviceToHost, streams[stream]));
			}
		}
		//�����豸�ڴ�
		//if (patch_size != _patch_size)cuda_free();
	}
	cuda_free();
	for (int stream = 0; stream < _stream_num; ++stream)
	{
		CUFFT_CUDA_ERROR(hipStreamSynchronize(streams[stream]));
	}
	for (int stream = 0; stream < _stream_num; stream++)
	{
		// Destroy streams.
		CUFFT_CUDA_ERROR(hipStreamDestroy(streams[stream]));
		CUFFT_CUDA_ERROR(hipfftDestroy(fftPlanFwd[stream]));
	}
}

void SignalProcessCUDA::cuda_free()
{
	CUDA_FREE(_d_dataline);
	CUDA_FREE(_d_dataline_comp);
	CUDA_FREE(_d_dft_result_comp);
	CUDA_FREE(_d_signal_real);
	CUDA_FREE(_d_signal_abs);
	CUDA_FREE(_d_idft_result_comp);
}

signal_process_model<float>* signal_process_GPU(unsigned int height,unsigned int width,unsigned int num, unsigned int patch_num, unsigned int stream_num)
{
	return new SignalProcessCUDA(height, width, num, patch_num, stream_num);
}