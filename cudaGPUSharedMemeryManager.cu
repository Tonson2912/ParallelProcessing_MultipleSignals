#include "hip/hip_runtime.h"
#include ""
#include"func_header.h"

template<typename T>
class cuda_pinned_memery_manager :public cuda_memery_manager<T>
{
public:
	bool mallocMemery(void** memery_ptr, unsigned long long size)
	{
		return hipHostMalloc(memery_ptr, size) == hipSuccess;
	}
	void freeMemery(void* memery_ptr)
	{
		if (memery_ptr == nullptr)return;
		if (hipHostFree(memery_ptr) != hipSuccess)
		{
			cout << "Non-GPU memory" << endl;
			return;
		}
		memery_ptr = nullptr;
	}
};

cuda_memery_manager<float>* memery_manager_GPU()
{
	return new cuda_pinned_memery_manager<float>();
}